#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hapi.h"

__global__ void helloKernel() {}

void invokeKernel(hipStream_t stream, void* cb) {
  //hapiEnqueue(wr);
  helloKernel<<<dim3(1, 1), dim3(1, 1), 0, stream>>>();
  //hapiAddCallback(stream, cb);
  hipDeviceSynchronize();
}
